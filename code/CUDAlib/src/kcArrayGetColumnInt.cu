//#include "cuda/hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include <hiprand.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#include "mex.h"

#include "kcDefs.h"
#include "kcArrayFunctions.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {



    mwSize *size  = (mwSize *)mxGetPr(mxGetField(prhs[0],0,KC_ARRAY_SIZE));

    int *d_a;
    d_a = kcGetArrayDataInt(prhs[0]);

    mwSize *size2 = (mwSize*)malloc(sizeof(mwSize)*2);
    size2[0] = size[0];
    size2[1] = 1; 


    int cNum = (int)mxGetScalar(prhs[1]);

    if(cNum < size[1]) {
        plhs[0] = kcSetupEmptyArray(2,size2);
        unsigned KC_PTR_SIZE int * ptr = (unsigned KC_PTR_SIZE int*)mxGetPr(mxGetField(plhs[0],0,KC_ARRAY_PTR));
        *ptr = (unsigned KC_PTR_SIZE int)(&(d_a[cNum*(size[0])]));
        
        unsigned int * type = (unsigned int*)mxGetPr(mxGetField(plhs[0],0,KC_ARRAY_TYPE));
        *type = KC_INT_ARRAY;
    }
    else {
        plhs[0] = mxCreateNumericMatrix(1,1,mxDOUBLE_CLASS,mxREAL);
        mexPrintf("Index out-of-bounds\n");
    }
}
