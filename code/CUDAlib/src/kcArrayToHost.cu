//#include "cuda/hip/hip_runtime.h"
#include <math.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "mex.h"

#include "kcDefs.h"


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    if(nrhs != 1) {
        mexPrintf("Incorrect RHS args: expected 1 and received %d (kcArrayToHost) ", nrhs);
        mexErrMsgTxt("CUDA errors");
    }
    if(nlhs != 1) {
        mexPrintf("Incorrect LHS args: expected 1 and received %d (kcArrayToHost) ", nlhs);
        mexErrMsgTxt("CUDA errors");
    }

	//init data crap
    unsigned int mSize = (unsigned int)mxGetScalar(mxGetField(prhs[0],0,KC_ARRAY_NUMEL));
    unsigned int ndims = (unsigned int)mxGetScalar(mxGetField(prhs[0],0,KC_ARRAY_NDIM));
    const mwSize* size = (const mwSize*)mxGetPr(mxGetField(prhs[0],0,KC_ARRAY_SIZE));


    unsigned int memSize = mSize*sizeof(KC_FP_TYPE);
    KC_FP_TYPE *d_a;
    d_a = (KC_FP_TYPE *)(unsigned KC_PTR_SIZE int)mxGetScalar(mxGetField(prhs[0],0,KC_ARRAY_PTR));
    if(d_a == KC_NULL_ARRAY) {
        mexPrintf("Invalid GPU array\n");
        return;
    }

	
    plhs[0] = mxCreateNumericArray(ndims,size,KC_FP_TYPE_MATLAB,mxREAL);
    KC_FP_TYPE* ans = (KC_FP_TYPE*)mxGetData(plhs[0]);
    hipError_t copyResult = hipMemcpy(ans,d_a,memSize,hipMemcpyDeviceToHost);
    
    if(copyResult == hipErrorInvalidValue) {
        mexPrintf("hipErrorInvalidValue\n");
    }
    else if(copyResult == hipErrorInvalidDevicePointer) {
        mexPrintf("hipErrorInvalidDevicePointer\n");
    }
    else if(copyResult == hipErrorInvalidMemcpyDirection) {
        mexPrintf("hipErrorInvalidMemcpyDirection\n");
    }



}
