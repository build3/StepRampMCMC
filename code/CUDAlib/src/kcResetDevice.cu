#include <math.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>




#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#include "mex.h"

#include "kcDefs.h" //see for info on anything starting with KC_
#include "kcArrayFunctions.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    hipError_t ce;
    /*ce = hipSetDevice(0);
    if(ce != hipSuccess) {
        mexPrintf("Error selecting device: %d\n", (int)ce);
    }
    else {*/
    hipGetLastError();
        ce = hipDeviceReset();
        if(ce != hipSuccess) {
            mexPrintf("Error reseting device: %d\n", (int)ce);
        }
        else {
            mexPrintf("Device reset.\n");
        }
    //}
}
