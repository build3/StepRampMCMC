#include "mex.h"
#include <hip/hip_runtime.h>
#include "kcDefs.h" //see for info on anything starting with KC_
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    hipError_t ce;
    
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipSetDevice(deviceIndex);   
        if(ce == hipSuccess) {
            mexPrintf("Selected CUDA device (%d)\n", deviceIndex);
            return;
        }
    }

    mexPrintf("Unable to select any devices");

}
