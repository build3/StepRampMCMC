#include "mex.h"
#include <hip/hip_runtime.h>
#include "kcDefs.h" //see for info on anything starting with KC_
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    int currentDevice, newDevice;
    hipError_t ce;
    hipGetDevice(&currentDevice);
    
    mexPrintf("Current GPU device: %d\n",currentDevice);
    
    if(nrhs == 0) {
        ce = hipSetDevice(KC_GPU_DEVICE);
    }
    else {
        ce = hipSetDevice((int)mxGetScalar(prhs[0]));
    }
    if(ce != hipSuccess) {
        mexPrintf("Error selecting device ");
        mexPrintf(hipGetErrorString(ce));
        mexPrintf(" (%d)\n", (int)ce);
        mexErrMsgTxt("CUDA Errors");
    }

    
    hipGetDevice(&newDevice);
    mexPrintf("Changed to GPU device: %d\n",newDevice);

}
