#include "mex.h"
#include <hip/hip_runtime.h>
#include "kcDefs.h" //see for info on anything starting with KC_
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    hipError_t ce;
    
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        ce = hipSetDevice(deviceIndex);   
        if(ce == hipSuccess) {
            mexPrintf("Selected CUDA device %d\n", deviceIndex);
            return;
        }else{
            mexPrintf("Error selecting device %d ", deviceIndex);
            mexPrintf(hipGetErrorString(ce));
            mexPrintf(" (%d)\n", (int)ce);
            mexErrMsgTxt("CUDA Errors");

        }
    }

    mexPrintf("Unable to select any devices");

}
